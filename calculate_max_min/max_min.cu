#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <opencv2/opencv.hpp>
#include <iostream>
using namespace std;
using namespace cv;


/*
 * 	dim3 threadsPerBlock(40, 20);
 *	dim3 blockPerGrid(25, 200);

dim3 |     二维grid            |     二维block           |
     |   gridDim.x=25         |    blockDim.x=40       |
     |   gridDim.y=200        |    blockDim.y=20       |
     |   gridDim.z=1          |    blockDim.z=1        |
     |                        |                        |
     |   blockIdx.x=[0,24]    |    threadIdx.x=[0,39]  |
     |   blockIdx.y=[0,199]   |    threadIdx.y=[0,19]  |
     |   blockIdx.z=0         |    threadIdx.z=0       |
*/
__global__ void matSum(uchar *dataIn, int *dataOutSum, int *dataOutMax, int *dataOutMin, int imgHeight, int imgWidth)
{
	//__shard__ int _data[1600];
	const int number = 2048;

	extern __shared__ int _sum[]; //小图像块中求和共享数组
	__shared__ int _max[number];  //小图像块中求最大值共享数组
	__shared__ int _min[number];  //小图像块中求最小值共享数组

	int thread = threadIdx.x + threadIdx.y * blockDim.x; //一个block中所有thread的索引值
	int threadIndex = threadIdx.x + threadIdx.y * imgWidth; //每个小块中存放数据的thread索引值

	//每个小块中存放数据的block索引值
	int blockIndex1 = blockIdx.x * blockDim.x + 2 * blockIdx.y * blockDim.y * imgWidth; //40*20的上半block索引值
	int blockIndex2 = blockIdx.x * blockDim.x + (2 * blockIdx.y + 1) * blockDim.y * imgWidth; //40*20的下半block索引值

	int index1 = threadIndex + blockIndex1; //每个block中上半部分索引值
	int index2 = threadIndex + blockIndex2; //每个block中上半部分索引值

	//将待计算的40*40小图像块中的所有像素分两次传送到共享数组
	_sum[thread] = dataIn[index1]; //将上半部分的40*20中所有数据赋值到共享数组中
	_sum[thread + blockDim.x * blockDim.y] = dataIn[index2]; //将下半部分的40*20中所有数据赋值到共享数组中

	_max[thread] = dataIn[index1];
	_max[thread + blockDim.x * blockDim.y] = dataIn[index2];

	_min[thread] = dataIn[index1];
	_min[thread + blockDim.x * blockDim.y] = dataIn[index2];

	//利用归约算法求出40*40小图像块中1600个像素值中的和、最大值、最小值
	for (unsigned int s = number / 2; s > 0; s >>= 1)
	{
		if (thread < s)
		{
			_sum[thread] += _sum[thread + s];
			if (_max[thread] < _max[thread + s])
			{
				_max[thread] = _max[thread + s];
			}
			if (_min[thread] > _min[thread + s])
			{
				_min[thread] = _min[thread + s];
			}
		}
		__syncthreads(); //所有线程保持同步
	}
	if (threadIndex == 0)
	{
		//将每个小块中的结果储存到输出中
		dataOutSum[blockIdx.x + blockIdx.y * gridDim.x] = _sum[0];
		dataOutMax[blockIdx.x + blockIdx.y * gridDim.x] = _max[0];
		dataOutMin[blockIdx.x + blockIdx.y * gridDim.x] = _min[0];
	}
}

int main()
{
	Mat image = imread("test.jpg", 0);
	int sum[5000];
	int max[5000];
	int min[5000];
	imshow("src", image);

	size_t memSize = image.cols*image.rows*sizeof(uchar); //定义GPU图像分配内存大小
	int size = 5000 * sizeof(int); //定义GPU整型数组分配内存大小

	uchar *d_src = NULL; //定义设备端指针变量
	int *d_sum = NULL;
	int *d_max = NULL;
	int *d_min = NULL;

	hipMalloc((void**)&d_src, memSize); //申请GPU图像的GPU内存
	hipMalloc((void**)&d_sum, size);  //申请GPU整型数组的GPU内存
	hipMalloc((void**)&d_max, size);  //申请GPU整型数组的GPU内存
	hipMalloc((void**)&d_min, size);  //申请GPU整型数组的GPU内存

	hipMemcpy(d_src, image.data, memSize, hipMemcpyHostToDevice);//从主机端复制图片数据到设备端，复制大小为长×宽×uchar类型

	int imgWidth = image.cols;  //图片列数，即图像宽
	int imgHeight = image.rows; //图片行数，即图像高

	dim3 threadsPerBlock(40, 20); //cuda三维变量类型，每个block大小为40×20
	dim3 blockPerGrid(25, 200); //将8000×1000的图片分为25×200个小图像块

	double time0 = static_cast<double>(getTickCount());            //计时器开始
	matSum<<<blockPerGrid, threadsPerBlock, 4096*sizeof(int)>>> (d_src, d_sum, d_max, d_min, imgHeight, imgWidth);
	time0 = ((double)getTickCount() - time0) / getTickFrequency(); //计时结束
    cout << "The Run Time is :" << time0 << "s" << endl; //打印并行算法计算时间

    hipMemcpy(sum, d_sum, size, hipMemcpyDeviceToHost); //复制GPU变量计算结果返回主机端
    hipMemcpy(max, d_max, size, hipMemcpyDeviceToHost);
    hipMemcpy(min, d_min, size, hipMemcpyDeviceToHost);

    cout << "The sum is :" << sum[0] << endl;
    cout << "The max is :" << max[0] << endl;
    cout << "The min is :" << min[0] << endl;

    waitKey(0);

    hipFree(d_src); //释放GPU变量内存
    hipFree(d_sum);
    hipFree(d_max);
    hipFree(d_min);

    return 0;
}
